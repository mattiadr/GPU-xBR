#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/videoio.hpp>
#include <iostream>

#include "../common/img_utils.h"
#include "../common/xBR_utils.h"


__global__ void expand_pixel_kernel(unsigned int rows, unsigned int cols, PixelRGB *inputRGB, PixelYUV *inputYUV, PixelRGB *output, unsigned int scaleFactor) {
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	if (col >= cols || row >= rows)
		return;

	expand_pixel(rows, cols, inputRGB, inputYUV, output, scaleFactor, row, col);
}

void expand_frame(unsigned int rows, unsigned int cols, PixelRGB *d_input, PixelYUV *d_yuv_data, PixelRGB *d_output, unsigned int scaleFactor) {
	dim3 threadsPerBlock = dim3(min(1024, rows * cols));
	dim3 blocks = dim3(ceil(rows * cols / (float)threadsPerBlock.x));
	
	printf("Creating (%d, %d) blocks with (%d, %d) threads...\n", blocks.x, blocks.y, threadsPerBlock.x, threadsPerBlock.y);
	rgb_to_yuv_kernel<<<blocks, threadsPerBlock>>>(rows * cols, d_input, d_yuv_data);
	hipError_t err = hipDeviceSynchronize();

    if (err != hipSuccess){
        printf("Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // TODO: reduce number of registers for the kernel
    threadsPerBlock = dim3(min(cols, 32), min(rows, 32));
	blocks = dim3(ceil(cols / (float)threadsPerBlock.x), ceil(rows / (float)threadsPerBlock.y));

	printf("Creating (%d, %d) blocks with (%d, %d) threads...\n", blocks.x, blocks.y, threadsPerBlock.x, threadsPerBlock.y);
	expand_pixel_kernel<<<blocks, threadsPerBlock>>>(rows, cols, d_input, d_yuv_data, d_output, scaleFactor);
    std::cout << hipGetLastError() << std::endl;

	err = hipDeviceSynchronize();

    if (err != hipSuccess){
        printf("Failed to launch kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

}

void expand_image(std::string input_path, std::string output_path, unsigned int scaleFactor) {
	cv::Mat img = cv::imread(input_path, cv::IMREAD_COLOR);
	PixelRGB *rgb_data = (PixelRGB *) img.data;
	PixelRGB *output = (PixelRGB *) malloc(img.rows * img.cols * scaleFactor * scaleFactor * sizeof (PixelRGB));

	PixelRGB *d_rgb_data, *d_output;
	PixelYUV *d_yuv_data;
	hipMalloc((void**)&d_rgb_data, (img.rows * img.cols) * sizeof(PixelRGB));
	hipMalloc((void**)&d_output, (img.rows * img.cols * scaleFactor * scaleFactor) * sizeof(PixelRGB));
	hipMalloc((void**)&d_yuv_data, (img.rows * img.cols) * sizeof(PixelYUV));

	hipMemcpy(d_rgb_data, rgb_data, (img.rows * img.cols) * sizeof(PixelRGB), hipMemcpyHostToDevice);

	expand_frame(img.rows, img.cols, d_rgb_data, d_yuv_data, d_output, scaleFactor);

	hipMemcpy(output, d_output, (img.rows * img.cols * scaleFactor * scaleFactor) * sizeof(PixelRGB), hipMemcpyDeviceToHost);

	cv::Mat img_out(img.rows * scaleFactor, img.cols * scaleFactor, CV_8UC3, (void *) output);
	cv::imwrite(output_path, img_out);
}

void expand_video(std::string input_path, std::string output_path, unsigned int scaleFactor) {
	cv::VideoCapture video(input_path);

	if (!video.isOpened()) {
		std::cerr << "Error opening video file" << std::endl;
		return;
	}

	double fps = video.get(cv::CAP_PROP_FPS);
	double frame_width = video.get(cv::CAP_PROP_FRAME_WIDTH);
	double frame_heigth = video.get(cv::CAP_PROP_FRAME_HEIGHT);
	int fourcc = static_cast<int>(video.get(cv::CAP_PROP_FOURCC));

	PixelRGB *rgb_data;
	PixelRGB *output = (PixelRGB *) malloc(frame_heigth * frame_width * scaleFactor * scaleFactor * sizeof (PixelRGB));
	cv::VideoWriter out_video(output_path, fourcc, fps, cv::Size(frame_width * scaleFactor, frame_heigth * scaleFactor));

	cv::Mat frame;
	while (1) {
		video >> frame;
		if (frame.empty())
			break;
		rgb_data = (PixelRGB *) frame.data;

		// TODO
		// expand_frame(frame_heigth, frame_width, rgb_data, output, scaleFactor);

		cv::Mat frame_out = cv::Mat(frame_heigth * scaleFactor, frame_width * scaleFactor, CV_8UC3, (uchar*) output);
		
		out_video << frame_out;
	}
	video.release();
	out_video.release();
}

int main(int argc, char const *argv[]) {
	// TODO check number of parameters
	std::string input_path = argv[2];
	std::string output_path = argv[3];
	int scaleFactor = atoi(argv[1]);
	
	// expand_video(input_path, output_path, scaleFactor);
	expand_image(input_path, output_path, scaleFactor);

	return 0;
}
