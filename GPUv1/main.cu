#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <opencv2/videoio.hpp>
#include <iostream>
#include <cmath>

#include "img_utils.h"
#include "xBR_utils.h"

#include "video_capture.c"

#define BLOCK_DIM 32
#define TILE_DIM 28
#define OFFSET 2

__global__ void expand_pixel_kernel(const unsigned int rows, const unsigned int cols, const PixelRGB *input, PixelRGB *output, const unsigned int scaleFactor) {
	const int t_row = threadIdx.y;
	const int t_col = threadIdx.x;

	const int o_row = blockIdx.y * TILE_DIM + t_row;
	const int o_col = blockIdx.x * TILE_DIM + t_col;

	const int i_row = o_row - OFFSET;
	const int i_col = o_col - OFFSET;

	// declare shared memory
	__shared__ PixelRGB sh_inputRGB[BLOCK_DIM * BLOCK_DIM];
	__shared__ PixelYUV sh_inputYUV[BLOCK_DIM * BLOCK_DIM];

	// copy data in shared memory
	PixelRGB p;
	if (i_row >= 0 && i_row < rows && i_col >= 0 && i_col < cols) {
		p = input[i_row * cols + i_col];
	} else if (i_row < 0 && i_col < 0) {
		// top left corner
		p = input[0 * cols + 0];
	} else if (i_row < 0 && i_col >= cols) {
		// top right corner
		p = input[0 * cols + (cols - 1)];
	} else if (i_row >= rows && i_col < 0) {
		// bottom left corner
		p = input[(rows - 1) * cols + 0];
	} else if (i_row >= rows && i_col >= cols) {
		// bottom right corner
		p = input[(rows - 1) * cols + (cols - 1)];
	} else if (i_row < 0) {
		// top edge
		p = input[0 * cols + i_col];
	} else if (i_row >= rows) {
		// bottom edge
		p = input[(rows - 1) * cols + i_col];
	} else if (i_col < 0) {
		// left edge
		p = input[i_row * cols + 0];
	} else if (i_col >= cols) {
		// right edge
		p = input[i_row * cols + (cols - 1)];
	} else {
		// this should never happen
		assert(false);
	}

	__syncthreads();

	sh_inputRGB[t_row * blockDim.x + t_col] = p;
	sh_inputYUV[t_row * blockDim.x + t_col] = rgb_to_yuv(p);

	__syncthreads();

	if (t_row >= TILE_DIM || t_col >= TILE_DIM || o_row >= rows || o_col >= cols) {
		return;
	}

	expand_pixel_tiling(cols, BLOCK_DIM, sh_inputRGB, sh_inputYUV, output, scaleFactor, t_row + OFFSET, t_col + OFFSET, o_row, o_col);
}

void expand_frame(unsigned int rows, unsigned int cols, PixelRGB3 *d_input_3, PixelRGB3 *d_output_3, PixelRGB *d_input, PixelRGB *d_output, unsigned int scaleFactor) {
	// convert frame from 3 to 4 channels
	dim3 threadsPerBlock_convert(min(1024, rows * cols));
	dim3 blocks_convert(ceil(rows * cols / (float) threadsPerBlock_convert.x));

	rgb3_to_rgb_kernel<<<blocks_convert, threadsPerBlock_convert>>>(rows * cols, d_input_3, d_input);

	hipError_t err = hipDeviceSynchronize();

	if (err != hipSuccess){
		printf("Failed to launch kernel (error code %d) %s!\n", hipGetLastError(), hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// expand pixels
	dim3 threadsPerBlock_expand(BLOCK_DIM, BLOCK_DIM);
	dim3 blocks_expand(ceil(cols / (float) TILE_DIM), ceil(rows / (float) TILE_DIM));

	expand_pixel_kernel<<<blocks_expand, threadsPerBlock_expand>>>(rows, cols, d_input, d_output, scaleFactor);

	err = hipDeviceSynchronize();

	if (err != hipSuccess){
		printf("Failed to launch kernel (error code %d) %s!\n", hipGetLastError(), hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	// convert frame from 4 to 3 channels
	blocks_convert.x *= scaleFactor * scaleFactor;
	rgb_to_rgb3_kernel<<<blocks_convert, threadsPerBlock_convert>>>(rows * cols * scaleFactor * scaleFactor, d_output, d_output_3);

	err = hipDeviceSynchronize();

	if (err != hipSuccess){
		printf("Failed to launch kernel (error code %d) %s!\n", hipGetLastError(), hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
}

void expand_image(std::string input_path, std::string output_path, unsigned int scaleFactor) {
	cv::Mat img = cv::imread(input_path, cv::IMREAD_COLOR);

	PixelRGB3 *output = (PixelRGB3 *) malloc(img.rows * img.cols * scaleFactor * scaleFactor * sizeof (PixelRGB3));

	// allocate memory on device
	PixelRGB3 *d_input_3, *d_output_3;
	PixelRGB *d_input, *d_output;

	hipMalloc((void **) &d_input_3, (img.rows * img.cols) * sizeof(PixelRGB3));
	hipMalloc((void **) &d_output_3, (img.rows * img.cols * scaleFactor * scaleFactor) * sizeof(PixelRGB3));
	hipMalloc((void **) &d_input, (img.rows * img.cols) * sizeof(PixelRGB));
	hipMalloc((void **) &d_output, (img.rows * img.cols * scaleFactor * scaleFactor) * sizeof(PixelRGB));

	// copy data to device
	hipMemcpy(d_input_3, img.data, (img.rows * img.cols) * sizeof(PixelRGB3), hipMemcpyHostToDevice);

	expand_frame(img.rows, img.cols, d_input_3, d_output_3, d_input, d_output, scaleFactor);

	// copy data from device
	hipMemcpy(output, d_output_3, (img.rows * img.cols * scaleFactor * scaleFactor) * sizeof(PixelRGB3), hipMemcpyDeviceToHost);

	cv::Mat img_out(img.rows * scaleFactor, img.cols * scaleFactor, CV_8UC3, (void *) output);
	cv::imwrite(output_path, img_out);
}

void expand_video(std::string input_path, std::string output_path, unsigned int scaleFactor) {
	cv::VideoCapture video(input_path);

	if (!video.isOpened()) {
		std::cerr << "Error opening video file" << std::endl;
		return;
	}

	double fps = video.get(cv::CAP_PROP_FPS);
	double frame_width = video.get(cv::CAP_PROP_FRAME_WIDTH);
	double frame_heigth = video.get(cv::CAP_PROP_FRAME_HEIGHT);
	int fourcc = static_cast<int>(video.get(cv::CAP_PROP_FOURCC));

	PixelRGB *output = (PixelRGB *) malloc(frame_heigth * frame_width * scaleFactor * scaleFactor * sizeof (PixelRGB));
	cv::VideoWriter out_video(output_path, fourcc, fps, cv::Size(frame_width * scaleFactor, frame_heigth * scaleFactor));

	// allocate memory on device
	PixelRGB3 *d_input_3, *d_output_3;
	PixelRGB *d_input, *d_output;

	hipMalloc((void **) &d_input_3, (frame_heigth * frame_width) * sizeof(PixelRGB3));
	hipMalloc((void **) &d_output_3, (frame_heigth * frame_width * scaleFactor * scaleFactor) * sizeof(PixelRGB3));
	hipMalloc((void **) &d_input, (frame_heigth * frame_width) * sizeof(PixelRGB));
	hipMalloc((void **) &d_output, (frame_heigth * frame_width * scaleFactor * scaleFactor) * sizeof(PixelRGB));

	// process frames
	cv::Mat frame;
	while (1) {
		video >> frame;
		if (frame.empty())
			break;

		// copy data to device
		hipMemcpy(d_input_3, frame.data, (frame_heigth * frame_width) * sizeof(PixelRGB3), hipMemcpyHostToDevice);

		expand_frame(frame_heigth, frame_width, d_input_3, d_output_3, d_input, d_output, scaleFactor);

		// copy data from device
		hipMemcpy(output, d_output_3, (frame_heigth * frame_width * scaleFactor * scaleFactor) * sizeof(PixelRGB3), hipMemcpyDeviceToHost);

		cv::Mat frame_out = cv::Mat(frame_heigth * scaleFactor, frame_width * scaleFactor, CV_8UC3, (void *) output);

		// save frame to video
		out_video << frame_out;
	}
	video.release();
	out_video.release();
}

int main(int argc, char const *argv[]) {
	if (argc < 3) {
		std::cout << "USAGE - " << argv[0] << ": scaleFactor type [inputFile] [outputFile]" << std::endl;
		std::cout << "TYPES: (i)mage, (v)ideo, (c)apture" << std::endl;
		return 0;
	}

	int scaleFactor = atoi(argv[1]);
	std::string type = argv[2];
	
	if (type == "i" || type == "image") {
		if (argc < 5) {
			std::cerr << "Missing file" << std::endl;
			return 1;
		}
		std::string input_path = argv[3];
		std::string output_path = argv[4];
		expand_image(input_path, output_path, scaleFactor);
		return 0;
	} else if (type == "v" || type == "video") {
		if (argc < 5) {
			std::cerr << "Missing file" << std::endl;
			return 1;
		}
		std::string input_path = argv[3];
		std::string output_path = argv[4];
		expand_video(input_path, output_path, scaleFactor);
		return 0;
	} else if (type == "c" || type == "capture") {
		expand_capture(scaleFactor);
	}

	std::cerr << "Type not recognized" << std::endl;

	return 1;
}
